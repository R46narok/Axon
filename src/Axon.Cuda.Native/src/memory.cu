#include "memory.cuh"

void* allocate_global_memory(int64_t iBytes)
{
    void* ptr;
    F1_CUDA_ASSERT(hipMalloc(&ptr, iBytes));
    return ptr;
}

void destroy_global_memory(void* ptr)
{
    hipFree(ptr);
}

void copy_host_to_device(void* pSrc, void* pDst, int64_t iLength)
{
    hipMemcpy(pDst, pSrc, iLength, hipMemcpyHostToDevice);
}

void copy_device_to_host(void* pSrc, void* pDst, int64_t iLength)
{
    hipMemcpy(pDst, pSrc, iLength, hipMemcpyDeviceToHost);
}

void copy_device_to_device(void* pSrc, void* pDst, int64_t iLength)
{
    hipMemcpy(pDst, pSrc, iLength, hipMemcpyDeviceToDevice);
}

void device_memset(void* pDst, int64_t iLength, int value)
{
    hipMemset(pDst, value, iLength);
}
