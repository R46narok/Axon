//
// Created by Acer on 23.7.2022 г..
//

#include "core.cuh"

static ErrorCallback s_Callback;
void set_error_callback(ErrorCallback callback)
{
    s_Callback = callback;
}

void gpuAssert(hipError_t code, const char* file, int line)
{
    if (code != hipSuccess && s_Callback != nullptr)
    {
        s_Callback((int)code, hipGetErrorString(code), file, line);
    }
}
