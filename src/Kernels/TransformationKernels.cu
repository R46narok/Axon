#include "hip/hip_runtime.h"
//
// Created by Acer on 18.10.2022 г..
//
#include "TransformationKernels.cuh"
#include <hipblas.h>

#pragma comment(lib, "cublas.lib")
#define BLOCK_SIZE 16

namespace Axon
{
    static hipblasHandle_t s_Handle;

    void MatrixDotKernel(float* pFirst, float* pSecond, float* pOutput, int firstRows, int firstColumns, int secondColumns)
    {
        unsigned int grid_rows = ceil((float)firstRows / BLOCK_SIZE);
        unsigned int grid_cols = ceil((float)secondColumns / BLOCK_SIZE);

        dim3 dimGrid(grid_rows, grid_cols);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

//    multiply_kernel<<<dimGrid, dimBlock>>>((float*)pFirst, (float*)pSecond, (float*)pOutput,
//                                           firstRows, firstColumns, secondColumns);

        if (s_Handle == nullptr) hipblasCreate(&s_Handle);

        float alpha = 1.0f, beta = 0.0f;
        hipblasSgemm(s_Handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    firstRows, secondColumns, firstColumns,
                    &alpha,
                    pFirst, firstRows,
                    pSecond, firstColumns,
                    &beta,
                    pOutput, firstRows);
    }

    __global__ void MatrixTransposeKernel(float* input, float* output, int rows, int columns)
    {
        unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
        unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

        if (xIndex < rows && yIndex < columns)
        {
//        unsigned int index_in = yIndex + columns * xIndex;
//        unsigned int index_out  = xIndex + rows * yIndex;
            unsigned int index_in = yIndex * rows + xIndex;
            unsigned int index_out = xIndex * columns + yIndex;
            output[index_out] = input[index_in];
        }
    }
}