#include "hip/hip_runtime.h"
#include "sum.cuh"
#include "sm_60_atomic_functions.h"
#include <nvtx3/nvToolsExt.h>

__global__ void sum_kernel(float* pInput, float* pOutput, int iRows, int iColumns)
{
    float sum = 0.0;
    for (int i = 0; i < iRows; ++i)
    {
        for (int j = 0; j < iColumns; ++j)
            sum += pInput[i * iColumns + j];
    }

    *pOutput = sum;
}

void sum(void* pInput, void* pOutput, int iRows, int iColumns)
{
    nvtxRangePush(__FUNCTION__);

    dim3 grid(1, 1, 1);
    dim3 threads(1, 1, 1);

    sum_kernel<<<grid, threads>>>((float*)pInput, (float*)pOutput, iRows, iColumns);
    F1_CUDA_ASSERT(hipPeekAtLastError());
    nvtxRangePop();
}
